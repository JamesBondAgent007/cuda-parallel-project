#include "hip/hip_runtime.h"
// Basic parallel convolution (slowest)


#include <iostream>

#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>

#include <hip/hip_runtime.h>

using namespace std;
using namespace cv;

#define SE_WIDTH 7 // It's always odd
#define SE_RADIUS (SE_WIDTH - 1)/2
#define GRID_DIM 32

// Cuda error handler
static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
	if(err!=hipSuccess)
	{
		fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",msg,file_name,line_number,hipGetErrorString(err));
		std::cin.get();
		exit(EXIT_FAILURE);
	}
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)


// srcImg is the image with padding, dstImg is without padding
__global__ void basicDilation(uchar* srcImg , uchar* dstImg , int srcImgCols , int dstImgRows , int dstImgCols)
{

	const int tx = blockIdx.x * blockDim.x + threadIdx.x;
	const int ty = blockIdx.y * blockDim.y + threadIdx.y;

	if(ty >= dstImgRows || tx >= dstImgCols)return;

	uchar min = srcImg[(ty + SE_RADIUS) * srcImgCols + (tx + SE_RADIUS)]; // Selecting SE central element

	for(int i=0 ; i<SE_RADIUS ; i++)
	{
		for (int j=0 ; j<SE_RADIUS ; j++)
		{
			uchar current = srcImg[(ty+i) * srcImgCols + (tx+j)];
			if (current < min)
				min = current;
		}
	}

	dstImg[ty * dstImgCols + tx] = min;

};


__global__ void basicErosion(uchar* srcImg , uchar* dstImg , int srcImgCols , int dstImgRows , int dstImgCols)
{

	const int tx = blockIdx.x * blockDim.x + threadIdx.x;
	const int ty = blockIdx.y * blockDim.y + threadIdx.y;

	if(ty >= dstImgRows || tx >= dstImgCols)return;

	uchar max = srcImg[(ty + SE_RADIUS) * srcImgCols + (tx + SE_RADIUS)]; // Selecting SE central element

	for(int i=0 ; i<SE_RADIUS ; i++)
	{
		for (int j=0 ; j<SE_RADIUS ; j++)
		{
			uchar current = srcImg[(ty+i) * srcImgCols + (tx+j)];
			if (current > max)
				max = current;
		}
	}

	dstImg[ty * dstImgCols + tx] = max;

};


// Wrapper function: choice = 0 -> Dilation
Mat launchKernel(Mat& img , Mat& immergedImg , int choice)
{

	// Allocating stuff on GPU
	uchar* devImgPtr;
	uchar* devImmergedImgPtr;
	int imgSize = img.rows*img.cols*sizeof(uchar);
	int immergedImgSize = immergedImg.rows*immergedImg.cols*sizeof(uchar);

	SAFE_CALL(hipMalloc((void**)&devImgPtr , imgSize) , "CUDA Malloc Failed");

	SAFE_CALL(hipMalloc((void**)&devImmergedImgPtr , immergedImgSize) , "CUDA Malloc Failed");
	SAFE_CALL(hipMemcpy(devImmergedImgPtr , immergedImg.ptr() , immergedImgSize , hipMemcpyHostToDevice) , "CUDA Memcpy Host To Device Failed");

	// Launching kernel(s)
	// Mysteriously Dim3 is structured like this (cols , rows , depth)
	dim3 blockDim(GRID_DIM , GRID_DIM , 1); // Using max threads number
	dim3 gridDim((img.cols + blockDim.x - 1)/blockDim.x , (img.rows + blockDim.y - 1)/blockDim.y , 1);

	if(choice == 0)
	{
		// ------------------------------START TIMER HERE------------------------------------

		basicDilation<<<gridDim , blockDim>>>(devImmergedImgPtr ,
											  devImgPtr ,
											  immergedImg.cols ,
											  img.rows ,
											  img.cols);
	}
	else
	{
		basicErosion<<<gridDim , blockDim>>>(devImmergedImgPtr ,
				  	  	  	  	  	  	  	 devImgPtr ,
				  	  	  	  	  	  	  	 immergedImg.cols ,
				  	  	  	  	  	  	  	 img.rows ,
				  	  	  	  	  	  	  	 img.cols);
	}

	// Checking for Kernel launch errors and wait for Device job to be done.
	SAFE_CALL(hipDeviceSynchronize() , "Kernel Launch Failed");

	// ----------------------------------END TIMER HERE--------------------------------------

	// Retrieving result
	SAFE_CALL(hipMemcpy(img.ptr() , devImgPtr , imgSize ,hipMemcpyDeviceToHost) , "CUDA Memcpy Host To Device Failed");

	// Freeing device
	SAFE_CALL(hipFree(devImgPtr) , "CUDA Free Failed");
	SAFE_CALL(hipFree(devImmergedImgPtr) , "CUDA Free Failed");

	return img;

}
