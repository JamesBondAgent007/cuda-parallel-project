#include "hip/hip_runtime.h"
#include <iostream>

#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>

#include <hip/hip_runtime.h>

using namespace std;
using namespace cv;

// Cuda error handler
static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
	if(err!=hipSuccess)
	{
		fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",msg,file_name,line_number,hipGetErrorString(err));
		std::cin.get();
		exit(EXIT_FAILURE);
	}
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)

// srcImg is the image with padding, dstImg is without padding
__global__ void basicDilation(int* srcImg , int* dstImg , int srcImgCols , int dstImgRows , int dstImgCols ,
							  int SErows , int SEcols)
{

	int paddingTop = (SErows-1)/2; // SErows and SEcols are assumed odd
	int paddingLeft = (SEcols-1)/2;

	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;

	int min = srcImg[(ty + paddingTop) * srcImgCols + (tx + paddingLeft)];

	if(ty < dstImgRows && tx < dstImgCols) // Checking idle threads
	{
		for(int i=0 ; i<SErows ; i++)
		{
			for (int j=0 ; j<SEcols ; j++)
			{
				int current = srcImg[(ty+i) * srcImgCols + (tx+j)];
				if (current < min)
					min = current;
			}
		}
	}

	dstImg[ty * dstImgCols + tx] = min;

};

__global__ void basicErosion(int* srcImg , int* dstImg , int srcImgCols , int dstImgRows , int dstImgCols ,
							 int SErows , int SEcols)
{

	int paddingTop = (SErows-1)/2; // SErows and SEcols are assumed odd
	int paddingLeft = (SEcols-1)/2;

	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;

	int max = srcImg[(ty + paddingTop) * srcImgCols + (tx + paddingLeft)];

	if(ty < dstImgRows && tx < dstImgCols) // Checking idle threads
	{
		for(int i=0 ; i<SErows ; i++)
		{
			for (int j=0 ; j<SEcols ; j++)
			{
				int current = srcImg[(ty+i+paddingTop) * srcImgCols + (tx+j+paddingLeft)];
				if (current > max)
					max = current;
			}
		}
	}

	dstImg[ty * dstImgCols + tx] = max;

};

// Wrapper function: choice = 0 -> Dilation
void launchKernel(Mat& img , Mat& immergedImg , int SErows , int SEcols , int choice)
{

	// Allocating stuff on GPU
	int* devImgPtr;
	int* devImmergedImgPtr;
	int imgSize = img.rows*img.cols*sizeof(int);
	int immergedImgSize = immergedImg.rows*immergedImg.cols*sizeof(int);

	SAFE_CALL(hipMalloc((void**)&devImgPtr , imgSize) , "CUDA Malloc Failed");
	SAFE_CALL(hipMemcpy(devImgPtr , img.ptr() , imgSize , hipMemcpyHostToDevice) , "CUDA Memcpy Host To Device Failed");

	SAFE_CALL(hipMalloc((void**)&devImmergedImgPtr , immergedImgSize) , "CUDA Malloc Failed");
	SAFE_CALL(hipMemcpy(devImmergedImgPtr , immergedImg.ptr() , immergedImgSize , hipMemcpyHostToDevice) , "CUDA Memcpy Host To Device Failed");

	// Launching kernel(s)
	dim3 gridDim(ceil(img.rows/32.0) , ceil(img.cols/32.0) , 1);
	dim3 blockDim(32 , 32 , 1); // Using max thread number

	if(choice == 0)
	{
		basicDilation<<<gridDim , blockDim>>>(devImmergedImgPtr ,
											  devImgPtr ,
											  immergedImg.cols ,
											  img.rows ,
											  img.cols ,
											  SErows ,
											  SEcols);

		SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed");
	}
	else
	{
		basicErosion<<<gridDim , blockDim>>>(devImmergedImgPtr ,
				  	  	  	  	  	  	  	 devImgPtr ,
				  	  	  	  	  	  	  	 immergedImg.cols ,
				  	  	  	  	  	  	  	 img.rows ,
				  	  	  	  	  	  	  	 img.cols ,
				  	  	  	  	  	  	  	 SErows ,
				  	  	  	  	  	  	  	 SEcols);

		SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed");
	}

	SAFE_CALL(hipMemcpy(img.ptr() , devImgPtr , imgSize ,hipMemcpyDeviceToHost),"CUDA Memcpy Host To Device Failed");

	imshow("Processed Img" , img);
	waitKey(0);

	// Freeing device
	SAFE_CALL(hipFree(devImgPtr) , "CUDA Free Failed");
	SAFE_CALL(hipFree(devImmergedImgPtr) , "CUDA Free Failed");

}
