#include "hip/hip_runtime.h"
#include <iostream>

#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>

#include <hip/hip_runtime.h>

using namespace std;
using namespace cv;

// Cuda error handler
static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
	if(err!=hipSuccess)
	{
		fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",msg,file_name,line_number,hipGetErrorString(err));
		std::cin.get();
		exit(EXIT_FAILURE);
	}
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)

// srcImg is the image with padding, dstImg is without padding
__global__ void basicDilation(int* srcImg , int* dstImg , int srcImgCols , int dstImgRows , int dstImgCols ,
							  int SErows , int SEcols)
{

	int paddingTop = (SErows-1)/2; // SErows and SEcols are assumed odd
	int paddingLeft = (SEcols-1)/2;

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int min = srcImg[(y + paddingTop) * srcImgCols + (x + paddingLeft)];

	if(y < dstImgRows && x < dstImgCols) // See professor's slides to understand this check
	{
		for(int i=0 ; i<SErows ; i++)
		{
			for (int j=0 ; j<SEcols ; j++)
			{
				int current = srcImg[(y+i) * srcImgCols + (x+j)];
				if (current < min)
					min = current;
			}
		}
	}

	dstImg[y * dstImgCols + x] = min;

};

__global__ void basicErosion(int* srcImg , int* dstImg , int srcImgCols , int dstImgCols ,
							 int SErows , int SEcols)
{

	int paddingTop = (SErows-1)/2; // SErows and SEcols are assumed odd
	int paddingLeft = (SEcols-1)/2;

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int max = srcImg[(y + paddingTop) * srcImgCols + (x + paddingLeft)];

	for(int i=0 ; i<SErows ; i++)
	{
		for (int j=0 ; j<SEcols ; j++)
		{
			int current = srcImg[(y+i+paddingTop) * srcImgCols + (x+j+paddingLeft)];
			if (current > max)
				max = current;
		}
	}

	dstImg[y * dstImgCols + x] = max;

};

// Wrapper function: choice = 0 -> Dilation
void launchKernel(Mat& img , Mat& immergedImg , int SErows , int SEcols , int choice)
{

	// Allocating stuff on GPU
	int* devImgPtr;
	int* devImmergedImgPtr;
	int imgSize = img.rows*img.cols*sizeof(int);
	int immergedImgSize = immergedImg.rows*immergedImg.cols*sizeof(int);

	SAFE_CALL(hipMalloc((void**)&devImgPtr , imgSize) , "CUDA Malloc Failed");
	SAFE_CALL(hipMemcpy(devImgPtr , img.ptr() , imgSize , hipMemcpyHostToDevice) , "CUDA Memcpy Host To Device Failed");

	SAFE_CALL(hipMalloc((void**)&devImmergedImgPtr , immergedImgSize) , "CUDA Malloc Failed");
	SAFE_CALL(hipMemcpy(devImmergedImgPtr , immergedImg.ptr() , immergedImgSize , hipMemcpyHostToDevice) , "CUDA Memcpy Host To Device Failed");

	// Launching kernel(s)
	dim3 gridDim(ceil(immergedImg.rows/32.0) , ceil(immergedImg.cols/32.0) , 1);
	dim3 blockDim(32 , 32 , 1); // Using max thread number

	if(choice == 0)
	{
		basicDilation<<<gridDim , blockDim>>>(devImmergedImgPtr ,
											  devImgPtr ,
											  immergedImg.cols ,
											  img.rows ,
											  img.cols ,
											  SErows ,
											  SEcols);

		SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed");
	}
	else
	{
		basicErosion<<<gridDim , blockDim>>>(devImmergedImgPtr ,
				  	  	  	  	  	  	  	 devImgPtr ,
				  	  	  	  	  	  	  	 immergedImg.cols ,
				  	  	  	  	  	  	  	 img.cols ,
				  	  	  	  	  	  	  	 SErows ,
				  	  	  	  	  	  	  	 SEcols);

		SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed");
	}

	// Use devImgPtr here to display result

	SAFE_CALL(hipMemcpy(img.ptr() , devImgPtr , imgSize ,hipMemcpyDeviceToHost),"CUDA Memcpy Host To Device Failed");

	imshow("Processed Img" , img);
	waitKey(0);

	// Freeing device
	SAFE_CALL(hipFree(devImgPtr) , "CUDA Free Failed");
	SAFE_CALL(hipFree(devImmergedImgPtr) , "CUDA Free Failed");

}
